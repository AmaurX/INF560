#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// includes
#include <helper_functions.h> // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>	  // helper functions for CUDA error checking and initialization

#include <hip/hip_runtime.h>

extern "C"
{
	#include "cuda_filters.h"
	#include "main.h"
}

__global__ void kern_gray_filter(struct pixel* pixelTab, char* checkTab, int numPixels)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int nProc = blockDim.x * gridDim.x;
	//1D grid-striding loop
	while(i < numPixels)
	{
		checkTab[i]++;
		int moy = (pixelTab[i].r + pixelTab[i].g +  pixelTab[i].b) / 3;
		if (moy < 0)
			moy = 0;
		if (moy > 255)
			moy = 255;
		pixelTab[i].r = moy;
		pixelTab[i].g = moy;
		pixelTab[i].b = moy;
		i+=nProc;
	}
}

extern "C" int cuda_test()
{
	printf("cuda_test was succesfully linked\n");
	return 0;
}

extern "C" int cuda_gray_filter(struct pixel* pixelTab, int numPixels, int stream)
{
	struct pixel* d_pixelTab;
	size_t tabSize = numPixels * sizeof(struct pixel);
	// for debug purpose : check that each elt is processed once
	size_t checkTabSize = numPixels * sizeof(char);
	char* checkTab = (char*) calloc(numPixels, sizeof(char));
	char* d_checkTab;

	int nThreads = (int) min(1024, (int)sqrt(numPixels));
	int nBlocks = (int) min(65535, (int)(1+numPixels/nThreads));
	dim3 gridDim;
	gridDim.x = nBlocks;
	dim3 blockDim;
	blockDim.x = nThreads;

	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	checkCudaErrors(hipMalloc((void **)&d_pixelTab, tabSize));
	checkCudaErrors(hipMalloc((void **)&d_checkTab, checkTabSize));

	//initialize the device memory
	checkCudaErrors(hipMemcpy(d_pixelTab, pixelTab, tabSize,	hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_checkTab, checkTab, checkTabSize,	hipMemcpyHostToDevice));
	
	hipEventRecord(start, 0);
	kern_gray_filter<<<gridDim, blockDim>>>(d_pixelTab, d_checkTab, numPixels);
	hipEventRecord(end, 0);
	
	//copy back
	checkCudaErrors(hipMemcpy(pixelTab, d_pixelTab, tabSize,	hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(checkTab, d_checkTab, checkTabSize,	hipMemcpyDeviceToHost));
	
	// check each elt is treated only once
	int totalCheck = 0;
	for(int i = 0 ; i < numPixels ; i++)
	{
		int diff = abs(checkTab[i] -1);
		if(diff != 0)
		{
			dbprintf("pixl %d check failed : %d\n", i, checkTab[i]);
		}
		totalCheck += diff;
	}

	hipEventSynchronize(end);
	float duration;
	hipEventElapsedTime(&duration, start, end);

	dbprintf("gray check = %d in %.2f ms (%d)\n", totalCheck, duration, numPixels );

	return 0;

}
